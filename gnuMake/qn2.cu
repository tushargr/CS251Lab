
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<sys/time.h>
#include<string.h>
#include<math.h>

#define NUM 10000000

#define CUDA_ERROR_EXIT(str) do{\
                                    hipError_t err = hipGetLastError();\
                                    if( err != hipSuccess){\
                                             printf("Cuda Error: '%s' for %s\n", hipGetErrorString(err), str);\
                                             exit(-1);\
                                    }\
                             }while(0);
#define TDIFF(start, end) ((end.tv_sec - start.tv_sec) * 1000000UL + (end.tv_usec - start.tv_usec))


__global__ void calculate(long * gpu_mem,long nthreads,long pwr, long start, long end)
{
      int tid = blockDim.x * blockIdx.x + threadIdx.x;
      if(tid >= nthreads)
           return;
               
      if(tid%2==0){
      		gpu_mem[start+tid*pwr]=gpu_mem[start+tid*pwr]^gpu_mem[start+tid*pwr+pwr-1];
      }
      else{
      		gpu_mem[start+tid*pwr+pwr-1]=gpu_mem[start+tid*pwr]^gpu_mem[start+tid*pwr+pwr-1];
      }     
}

int main(int argc, char **argv)
{
    struct timeval t_start, t_end;
    long i;
    long *gpu_mem;   
    unsigned long num = NUM;   /*Default value of num from MACRO*/
    int SEED;
    long blocks;

    if(argc == 3){
         num = atoi(argv[1]);   /*Update after checking*/
         if(num <= 0)
               num = NUM;
	SEED=atoi(argv[2]);
    }
    unsigned long n=num;	
    /* Allocate host (CPU) memory and initialize*/
    srand(SEED);
    long * ar = (long *)malloc(num*sizeof(long));
    for(i=0; i<num; ++i){
    	*(ar+i)= (long)(random());
    }
   
    
    gettimeofday(&t_start, NULL);
    
    /* Allocate GPU memory and copy from CPU --> GPU*/

 
    	
	
    long start=0;
    long end=-1;
    long ans=0;
    while(1){
	hipMalloc(&gpu_mem, n * sizeof(long));
	CUDA_ERROR_EXIT("hipMalloc");
	hipMemcpy(gpu_mem, ar, n * sizeof(long) , hipMemcpyHostToDevice);
    	CUDA_ERROR_EXIT("hipMemcpy");

	if(num==0)break;
    	start=end+1;
    	long lg=floor(log(num)*1.00/log(2));
    	end=(long)(start+pow(2,lg)-1);
    	if(num==1){
    		ans=(ans^ar[start]);
    		break;
    	}
	
    	long j;
    	long size=end+1-start;
    	long times=(long)(log(size)/log(2));
    	for(j=1;j<=times;j++){
    		long nthreads=size/((long)(pow(2,j)));
    		long pwr=(long)(pow(2,j)); 
    		blocks=nthreads/1024;
    		if(nthreads%1024)blocks++;
    		calculate<<<blocks, 1024>>>(gpu_mem,nthreads,pwr,start,end); 		
    		CUDA_ERROR_EXIT("kernel invocation");
    	}
    	hipMemcpy(ar, gpu_mem, n * sizeof(long) , hipMemcpyDeviceToHost);
    	CUDA_ERROR_EXIT("memcpy");
	hipFree(gpu_mem);

    	ans=(ans^ar[start]);
    	num=num-(long)(pow(2,lg));
    }
    
    gettimeofday(&t_end, NULL);
    
    printf("Total time = %ld microsecs\n", TDIFF(t_start, t_end));
    hipFree(gpu_mem);
   
    printf("result = %ld\n", ans);
    free(ar);
}
 
